#include "hip/hip_runtime.h"
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
	#ifdef CUDA_ERROR_CHECK
		if (hipSuccess != err)
		{
			fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
				file, line, hipGetErrorString(err));
			exit(-1);
		}
	#endif

    return;
}

inline void __cudaCheckError(const char *file, const int line)
{
	#ifdef CUDA_ERROR_CHECK
		hipError_t err = hipGetLastError();
		if (hipSuccess != err)
		{
			fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
				file, line, hipGetErrorString(err));
			exit(-1);
		}

		err = hipDeviceSynchronize();
		if (hipSuccess != err)
		{
			fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
				file, line, hipGetErrorString(err));
			exit(-1);
		}
	#endif

    return;
}

__global__ void rgb_2_grey(uchar* const greyImage, const uchar4* const rgbImage, int rows, int columns)
{
    int rgb_x = blockIdx.x * blockDim.x + threadIdx.x; //x coordinate of pixel
    int rgb_y = blockIdx.y * blockDim.y + threadIdx.y; //y coordinate of pixel

    if ((rgb_x >= columns) && (rgb_y >= rows)) {
        return;
    }

    int rgb_ab = rgb_y*columns + rgb_x; //absolute pixel position
    uchar4 rgb_Img = rgbImage[rgb_ab];
    greyImage[rgb_ab] = uchar((float(rgb_Img.x))*0.299f + (float(rgb_Img.y))*0.587f + (float(rgb_Img.z))*0.114f);
}

using namespace cv;
using namespace std;

void Proc_Img(uchar4** h_RGBImage, uchar** h_greyImage, uchar4 **d_RGBImage, uchar** d_greyImage);
void RGB_2_Greyscale(uchar* const d_greyImage, uchar4* const d_RGBImage, size_t num_Rows, size_t num_Cols);
void Save_Img();

Mat img_RGB;
Mat img_Grey;
uchar4 *d_rgbImg;
uchar *d_greyImg; 

int main()
{
        uchar4* h_rgbImg;
        //uchar4* d_rgbImge=0;
        uchar* h_greyImg;
        //uchar* d_greyImge=0;

        Proc_Img(&h_rgbImg, &h_greyImg, &d_rgbImg, &d_greyImg);
        RGB_2_Greyscale(d_greyImg, d_rgbImg, img_RGB.rows, img_RGB.cols);
        Save_Img();

    return 0;
}

void Proc_Img(uchar4** h_RGBImage, uchar** h_greyImage, uchar4 **d_RGBImage, uchar** d_greyImage){
    hipFree(0);
    CudaCheckError();

    //loads image into a matrix object along with the colors in BGR format (must convert to rgb).
    Mat img = imread("cinque_terre_small.jpg", CV_LOAD_IMAGE_COLOR);
    if (img.empty()){
        cerr << "couldnt open file ..." << "cinque_terre_small.jpg" << endl;
        exit(1);
    }

    //converts color type from BGR to RGB
    cvtColor(img, img_RGB, CV_BGR2RGBA);

    //allocate memory for new greyscale image. 
    //img.rows returns the range of pixels in y, img.cols returns range of pixels in x
    //CV_8UC1 means 8 bit unsigned(non-negative) single channel of color, aka greyscale.
    //all three of the parameters allow the create function in the Mat class to determine how much memory to allocate
    img_Grey.create(img.rows, img.cols, CV_8UC1);

    //creates rgb and greyscale image arrays
    *h_RGBImage = (uchar4*)img_RGB.ptr<uchar>(0); //.ptr is a method in the mat class that returns a pointer to the first element of the matrix.
    *h_greyImage = (uchar*)img_Grey.ptr<uchar>(0);        //this is just like a regular array/pointer mem address to first element of the array. This is templated
                                                          //in this case the compiler runs the function for returning pointer of type unsigned char. for rgb image it is
                                                          //cast to uchar4 struct to hold r,g, and b values.

    const size_t num_pix = (img_RGB.rows) * (img_RGB.cols); //amount of pixels 

    //allocate memory on gpu
    hipMalloc(d_RGBImage, sizeof(uchar4) * num_pix); //bites of 1 uchar4 times # of pixels gives number of bites necessary for array
    CudaCheckError();
    hipMalloc(d_greyImage, sizeof(uchar) * num_pix);//bites of uchar times # pixels gives number of bites necessary for array
    CudaCheckError();
    hipMemset(*d_greyImage, 0, sizeof(uchar) * num_pix);
    CudaCheckError();


    //copy array into allocated space
    hipMemcpy(*d_RGBImage, *h_RGBImage, sizeof(uchar4)*num_pix, hipMemcpyHostToDevice);
    CudaCheckError();


    d_rgbImg = *d_RGBImage;
    d_greyImg = *d_greyImage; 
}


void RGB_2_Greyscale(uchar* const d_greyImage, uchar4* const d_RGBImage, size_t num_Rows, size_t num_Cols){

    const int BS = 16;
    const dim3 blockSize(BS, BS);
    const dim3 gridSize((num_Cols / BS) + 1, (num_Rows / BS) + 1); 

    rgb_2_grey <<<gridSize, blockSize>>>(d_greyImage, d_RGBImage, num_Rows, num_Cols);

    hipDeviceSynchronize(); CudaCheckError();

}



void Save_Img(){

    const size_t num_pix = (img_RGB.rows) * (img_RGB.cols);
    hipMemcpy(img_Grey.ptr<uchar>(0), d_greyImg, sizeof(uchar)*num_pix, hipMemcpyDeviceToHost);
    CudaCheckError();


    imwrite("result.jpg", img_Grey);

    hipFree(d_rgbImg);
    hipFree(d_greyImg);

}